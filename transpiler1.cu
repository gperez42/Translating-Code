#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define MAX_LINES 1024
#define MAX_LINE_LENGTH 256

__device__ void device_strcpy(char *dest, const char *src) {
    int i = 0;
    while (src[i] != '\0') {
        dest[i] = src[i];
        i++;
    }
    dest[i] = '\0';  // Null-terminate the string
}

// Kernel to process the lines on the GPU
__global__ void processLineKernel(char *d_lines, int *d_flags, char *d_output, int numLines) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < numLines) {
        // Process variable declarations or print statements based on flags
        if (d_flags[idx] == 1) {  // Variable declaration
            device_strcpy(d_output + idx * MAX_LINE_LENGTH, d_lines + idx * MAX_LINE_LENGTH);
        } else if (d_flags[idx] == 2) {  // Print statement
            // Add handling for print statements here (e.g., printf)
        } else if (d_flags[idx] == 3) {  // Assignment operation
            device_strcpy(d_output + idx * MAX_LINE_LENGTH, d_lines + idx * MAX_LINE_LENGTH);
 		}  else if (d_flags[idx] == 4) {  // Handle #include <stdio.h>
            device_strcpy(d_output + idx * MAX_LINE_LENGTH, d_lines + idx * MAX_LINE_LENGTH);
        } else if (d_flags[idx] == 5) {  // Unsupported line
            device_strcpy(d_output + idx * MAX_LINE_LENGTH, "Unsupported line\n");
        }
        
     }
}

// Host-side function to preprocess the lines
void processLine(const char *line, FILE *outputFile, int *flags, int idx) {
    if (strncmp(line, "int ", 4) == 0 || strncmp(line, "float ", 6) == 0 || 
        strncmp(line, "double ", 7) == 0 || strncmp(line, "String", 7) == 0) {
        flags[idx] = 1; // Mark this line as a variable declaration
    }
    else if (strncmp(line, "System.out.println", 18) == 0 || strncmp(line, "System.out.print", 16) == 0) {
        flags[idx] = 2; // Mark this line for print statement handling
    }
    else if (strchr(line, '=') != NULL) {
        flags[idx] = 3; // Mark for simple assignment handling
    }
    else if (strncmp(line, "import", 6) == 0 || strncmp(line, "", 0) == 0) {
        flags[idx] = 0; // Skip imports or empty lines
    }
    else if (strncmp(line, "#include <stdio.h>", 18) == 0) {
        flags[idx] = 4; // Mark this line as a special include line
    }
    else {
        flags[idx] = 5; // Unsupported line
    }
}

// Main function
int main(int argc, char *argv[]) {
    char lines[MAX_LINES][MAX_LINE_LENGTH];
    int flags[MAX_LINES];
    FILE *inputFile = fopen(argv[1], "r");
    FILE *outputFile = fopen(argv[2], "w");

    if (!inputFile || !outputFile) {
        fprintf(stderr, "Error opening file\n");
        return 1;
    }

    int numLines = 0;
    while (fgets(lines[numLines], sizeof(lines[numLines]), inputFile)) {
        processLine(lines[numLines], outputFile, flags, numLines);
        numLines++;
    }

     // Write the #include <stdio.h> line first
    fprintf(outputFile, "#include <stdio.h>\n");

    fclose(inputFile);

    // Prepare the data for CUDA
    char *d_lines, *d_output;
    int *d_flags;

    hipMalloc((void**)&d_lines, MAX_LINES * MAX_LINE_LENGTH * sizeof(char));
    hipMalloc((void**)&d_flags, MAX_LINES * sizeof(int));
    hipMalloc((void**)&d_output, MAX_LINES * MAX_LINE_LENGTH * sizeof(char));

    hipMemcpy(d_lines, lines, MAX_LINES * MAX_LINE_LENGTH * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_flags, flags, MAX_LINES * sizeof(int), hipMemcpyHostToDevice);

     // Launch the kernel with numLines blocks and threads per block
    int threadsPerBlock = 256;
    int blocks = (numLines + threadsPerBlock - 1) / threadsPerBlock;
    processLineKernel<<<blocks, threadsPerBlock>>>(d_lines, d_flags, d_output, numLines);

    // Check for kernel errors
    hipDeviceSynchronize();
    
    // Copy the processed output back to the host
    char output[MAX_LINES][MAX_LINE_LENGTH];
    hipMemcpy(output, d_output, MAX_LINES * MAX_LINE_LENGTH * sizeof(char), hipMemcpyDeviceToHost);

     // Write the processed lines to the output file
    for (int i = 0; i < numLines; i++) {
        if (strlen(output[i]) > 0) {
            fprintf(outputFile, "%s\n", output[i]);
        }
    }

       // Clean up
    hipFree(d_lines);
    hipFree(d_flags);
    hipFree(d_output);

    fclose(outputFile);
    return 0;
}
